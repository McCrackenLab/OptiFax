#include "hip/hip_runtime.h"
/* 
 *OPO_CUDA_ODE23.cu
 *
 */

#include <hip/hip_runtime_api.h> 
#include <hip/hip_math_constants.h>
#include <hipfft/hipfft.h>
#include "OPO_ODE23_HEADER.hpp"
#include <hip/hip_runtime.h>
#include ""
#include <limits>

/*Define block size.*/
#define threadsPerBlock 256

/* Define imaginary constant */
#define I make_float2(0.0,1.0)
#define c0 make_float2(0.0,0.0)
//const float2 I = make_float2(0.0,1.0);
//const float2 c0 = make_float2(0.0,0.0);

__constant__ float w;
__constant__ float d_invdt2;  
__constant__ int Np = 1; 

// complex math functions
__device__ inline
float2 conjugate(float2 arg)
{
    return make_float2(arg.x, -arg.y);
}

__device__ inline
float2 complex_exp(float arg)
{
    return make_float2(cosf(arg), sinf(arg));
}

__device__ inline
float2 complex_pow(float2 arg, int n)
{
	// return make_float2(cosf(acosf(arg.x)*n), sinf(asinf(arg.y)*n));
	return make_float2(cosf(acosf(arg.x) * n), -sinf(asinf(-arg.y)*n));
}

__device__ inline
float2 complex_add(float2 const a, float2 b)
{
    return make_float2(a.x + b.x, a.y + b.y);
}

__device__ inline
float2 complex_minus(float2 a, float2 b)
{
    return make_float2(a.x - b.x, a.y - b.y);
}

__device__ inline
float2 complex_mult(float2 ab, float2 cd)
{
    return make_float2((ab.x * cd.x) - (ab.y * cd.y), (ab.x * cd.y) + (ab.y * cd.x));
}

__device__ inline
float2 complex_scale(float2 a, float s) 
{
	return make_float2(s * a.x, s * a.y);
}


__host__ __device__ inline
float fabsf2(const float2 &a) 
{ 
	return hypot( a.x , a.y );
	//return ; 
}


__device__ inline
float2 f2absf2(const float2 &a) 
{ 
	return make_float2(fabsf2(a),0.0); 
}


__global__
//__device__ inline 
 void complex_AddVec(	float2 * const A,
             			float2 const * const B,
                        float2 const * const C,
                        float const scale,
             		//	int const gi)
 						int const N)
{
	
    /* Calculate the global linear index, assuming a 1-d grid. */
    /*
    */
    int const gi = blockDim.x * blockIdx.x + threadIdx.x;
    if (gi < N) 
    {
	
        A[gi] = complex_add(B[gi] , complex_scale(C[gi],scale));
    }
}

 __global__ 
 void complex_ScaleVec(	float2 * const A,
             			float const b,
             			int const N,
             			//float2 const c0,
             			int const gi1,
             			int const gi2)
{
    /* Calculate the global linear index, assuming a 1-d grid. */
   	// 2D grid?
    int const gx = blockDim.x * blockIdx.x + threadIdx.x;
	int const gi = (blockDim.x * blockIdx.x) + threadIdx.x + (N * blockIdx.y);

    float pos, g, bc;
    //float a = 24.0;
	float a = 24.0;

    if (gx < N) 
    {
    	
    	if (gx > gi1 && gx < gi2) // Remove temporal aliasing     		
    	{
    		pos = (a*2.0/N) * (fabs( (float) gx - (N/2.0) ));
        	g = coshf(pos);
    		bc = (1 - (1/(g*g))) * b;
    		A[gi] = complex_scale(A[gi] , bc);
        	//A[gi] = complex_scale(A[gi] , 0.0*b);
    		//float bc = (cbrt(cbrt(cbrt(cbrt(cbrt( (float) pos)))))) * b;
    	}
    	else
    	{
    	
			A[gi] = complex_scale(A[gi] , b);
    	}
    	
    }
}
// Simple all in one NLFN kernel to start with.
/* NLfn = ((A0.^2).*exp(i(wt-(bdiffw0)*z)))
 * +(2*expi(-wt+(bdiffw0)*z).*abs(A0).^2)
 */
__global__
//__device__ inline 
void NLFN_kernel(float2 * const NL, 
				 float2 const * const Ax,
				 float2 const * const Ay,
				 double const dt,
				 float const w0, 
				 double bdwz,
				 //int gi) 
				 int Npoints)
{
	/* Calculate the global linear index, assuming a 1-d grid. */
	// 2D grid?
    int const gx = blockDim.x * blockIdx.x + threadIdx.x;
	int const gi = (blockDim.x * blockIdx.x) + threadIdx.x + (Npoints * blockIdx.y);
    if (gx < Npoints)
	/*
	for (int gi =  blockDim.x * blockIdx.x + threadIdx.x;
		gi < Npoints;
		gi += blockDim.x * gridDim.x)
	*/	
    {
    
		/* NL[gi] = ((A0[gi] * A0[gi]) * (expfi(wt[gi] - (bdiffw0 * z)))) +
    				(2 * expfi(-wt[gi] + (bdiffw0 * z)) * (abs(A0[gi]) * abs(A0[gi]) )); */

    	//float arg = ((w0 * t[gx]) - bdwz);
		float arg = ((w0 * (dt*gx)) - bdwz);

    	float2 AxAy		= complex_mult(Ax[gi],Ay[gi]);
    	float2 abAxAy 	= complex_mult(f2absf2(Ax[gi]),f2absf2(Ay[gi]));
    	float2 expon 	= complex_exp(arg);
    	float2 dbcnjex	= complex_add((conjugate(expon)),(conjugate(expon))); 
    	NL[gi] 			= complex_add(complex_mult(AxAy, expon), complex_mult(dbcnjex,abAxAy));
		// if (NL[gi].x == (FLT_MAX) || NL[gi].y == (FLT_MAX))
		// {
		// 	NL[gi] = make_float2(0.0,0.0);
		// }

    }

}

// Runge-Kutta KN Step Kernel
// K1=-h*1i*G*(NLfn-1/(2*dt)*(1i/w0)*(NLfn(ind1)-NLfn(ind2)))
__global__
//__device__ inline 
void KN_kernel (float2 const * const A0,
				float2 * const KN,
				float2 * const KN2,
				float2 * const NL,
				float const * const G,
				float const w0,
				float const h,
				int const k,
				float invdt2,
				//float2 const I,
				int Npoints,
				float2 * const AN,
				float b1,
                float b2)
{
	int in1, in2;

	/* Calculate the global linear index, assuming a 1-d grid. */
 	// 2D grid?
    int const gx = blockDim.x * blockIdx.x + threadIdx.x;
	int const gi = (blockDim.x * blockIdx.x) + threadIdx.x + (Npoints * blockIdx.y);
    
    int const d = 1;

	float2 iby2dtw0 = complex_mult(make_float2(invdt2/(d * w0), 0.0), I);
    float2 hiG = complex_mult(I, make_float2((-1.0 * h * G[k]),0.0));

    // if cell is not a boundary
    if (gx > d - 1 && gx < Npoints - d)
    {
		in1 = gi+d;
	    in2 = gi-d;
	}   
	else
	{
	    if (gx < d)
	    {
	    //in1 = gi+(2*d);
	    in1 = gi + d;	
	    in2 = Npoints - d + gi;
	    //in2 = gi;
	    }
	    if (gx > Npoints - (d + 1))
    	{
    	in1 = gi - (Npoints - d);
    	//in1 = gi;
    	//in2 = gi-(2*d);
    	in2 = gi - d;
	    }
	}

	float2 dNL = complex_minus(NL[in1], NL[in2]);
	KN[gi] = complex_mult(hiG, (complex_minus(NL[gi], complex_mult(iby2dtw0, dNL))));
	AN[gi] = complex_add(A0[gi],complex_add(complex_scale(KN[gi],b1),complex_scale(KN2[gi],b2)));
	// if (AN[gi].x == (FLT_MAX) || AN[gi].y == (FLT_MAX))
	// {
	// 	AN[gi] = make_float2(0.0,0.0);
	// }
	//AN[gi] = complex_add(A0[gi],complex_scale(complex_add(KN[gi],KN2[gi]),fracstep));
}


__global__ void Dispersion_kernel (	float2 * const E,
									float2 * const EFT,
									float const * const alpha_w,
									float const * const beta_op,
									int Npoints,
									int stepmod,
									int chunk,
									int Nchunks,
									float * const stepmods,
									int const k)									
{
	/* Calculate the global linear index, assuming a 1-d grid. */
    //int const gi = blockDim.x * blockIdx.x + threadIdx.x;
    //int sc;
    float2 bophmod = make_float2(1.0,0.0);
	//float2 bophmod;

	// 2D grid?
    int const gx = blockDim.x * blockIdx.x + threadIdx.x;
	int const gy = blockIdx.y;
	int const gi = (blockDim.x * blockIdx.x) + threadIdx.x + (Npoints * blockIdx.y);

	if (gx == k)
	{
		stepmods[k] = stepmod;
	}

    if (gx < Npoints)
    {
    	
    	// for (sc=0;sc<stepmod;sc++)
		// {
		// 	bophmod = complex_mult(bophmod,beta_op[gi]);
		// }
	
    	//bophmod = complex_pow(beta_op[gi],stepmod);

    	bophmod = complex_exp(-beta_op[gx] * stepmod);
		bophmod = complex_scale(bophmod, powf(alpha_w[gx],stepmod));
    	EFT[gi] = complex_mult(EFT[gi],bophmod);
/*
    	if (gi > 200 && gi < 3000)
    	{
        	EFT[gi] = complex_scale(EFT[gi] , 0.001);
    	}
*/
    	//E[chunk + (gi * Nchunks)] = EFT[gi];
        E[(gy * (Nchunks-1) * Npoints) + (chunk * Npoints) + gi] = EFT[gi];
    }

}
									

__global__ void Error_kernel (	float2 * const K1,
								float2 * const K2,
								float2 * const E,
								int k,
								int stepmod,
								int Npoints,
								float const maxpe,
								float const minpe,
								bool * max_min_flag)
{
	/* Calculate the global linear index, assuming a 1-d grid. */
    //int const gi = blockDim.x * blockIdx.x + threadIdx.x;
    float errfv, pcterrfv;

  	// 2D grid?
    int const gx = blockDim.x * blockIdx.x + threadIdx.x;
	int const gi = (blockDim.x * blockIdx.x) + threadIdx.x + (Npoints * blockIdx.y);
    if (gx < Npoints)
    {
    	if(fabsf2(E[gi]) > 0)
    	{
	    	errfv 	 = 0.5 * (fabsf2(complex_minus(K2[gi],K1[gi])));
			//errfv 	 =  0.01 * fabsf2(make_float2(1.0,0.0));
			//errfv 	 =  0.01;
	    	pcterrfv = 4 * 100.0 * (errfv / (fabsf2(E[gi])+1));
			//pcterrfv = 0.01;

	    	if (pcterrfv >= maxpe)
	    	{
	    		max_min_flag[0] = true;
	    	} 
	    	else
	    	{
		    	if (pcterrfv >= minpe)
		    	{
		    		max_min_flag[1] = true;
		    	} 
    		}
	 	}
    }

}


__global__ void copyKernel (float2 * Eft,
							float2 * ApFT,
							int Npoints,
							int chunk,
							int Nchunks)
{
	/* Calculate the global linear index, assuming a 1-d grid. */
    //int const gi = blockDim.x * blockIdx.x + threadIdx.x;

    //if (gi < Npoints)
	for (int gi =  blockDim.x * blockIdx.x + threadIdx.x;
		gi < Npoints;
		gi += blockDim.x * gridDim.x)	
    {
    	ApFT[chunk + (gi * Nchunks)] = Eft[gi];
    }

} 

// Host function called by MEX gateway.
 void OPO_TEST_CUDA(float2 * const d_E,
					float2 * const d_A0,
					float2 * const d_A1,
					float2 * const d_NL,
					float2 * d_K1,
					float2 * const d_K2,
                    float2 * const d_K3,
                    float2 * d_K4,    
				   	float const * const d_G,
					float const * const d_alpha_w,
				   	float const w0, 
				   	double const bdiffw0,
				   	float const h,
				   	int const Nsteps,
				   	int Npoints,
					int Nbatches,
				   	double const dt,
					float const * const d_beta_op,
					float const max_pct_err,
					float const min_pct_err,
					int const sel,
					float2 * const d_Ap,
					float * const d_stepmods)
{	
	// GPU variables
	float2* d_temp;
	float stepmod, stepend, stepprev, k, invdt2, invSize, hmod;
	double bdwz;
	int  chunk, Nchunks, gi1, gi2;
	bool* max_min_err_exc;

	// Precompute scalars:
	invdt2 	= 1.0 / (2.0 * dt);
	invSize = 1.0 / Npoints;
	k = 0.0;
	stepprev = 4.0;
	Nchunks = Nsteps / sel;
	// Constants defining cut off for temporal aliasing reduction
	gi1 = 6.0*(Npoints/16);
	gi2 = 10.0*(Npoints/16);

	hipHostMalloc(&max_min_err_exc, 2*sizeof(bool));

	// CUFFT plan simple API
  	hipfftHandle plan;
  	hipfftPlan1d(&plan, Npoints, HIPFFT_C2C, Nbatches);
	//hipfftPlanMany(&plan, 1, Npoints, HIPFFT_C2C, 1);

	//dim3 const blocksPerGrid(((Npoints) + threadsPerBlock - 1) / threadsPerBlock, Nbatches);
    dim3 const blocksPerGrid(Npoints / threadsPerBlock, Nbatches);
	//err_reset_kernel<<<1, 1 >>>();

    // Compute scalars
	const float butcher_as[] = {1.0/2.0, 3.0/4.0, 4.0/9.0, 1.0/3.0, 2.0/9.0};
	//const float butcher_as[] = {1.0/2.0, 1.0/4.0, 0.0, 1.0/4.0, 1.0/4.0};
	const float butcher_bs[] = {1.0/8.0, 1.0/3.0, 1.0/4.0, 7.0/24.0};
	//const float butcher_bs[] = {1.0/4.0, 1.0/4.0, 1.0/4.0, 1.0/4.0};
    stepmod = stepprev;
	hmod = stepmod * h;
    bdwz = bdiffw0 * k * h;

    NLFN_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_NL,d_A0,d_A0,dt,w0,bdwz,Npoints);
	//KN_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A0,d_K1,d_K1,d_NL,d_G,w0,hmod,k,invdt2,Npoints,d_A1,0.5,0.0);
	KN_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A0,d_K1,d_K1,d_NL,d_G,w0,hmod,k,invdt2,Npoints,d_A1,butcher_as[0],0.0);

	for (chunk = 0; chunk < Nchunks; chunk++)
	{
		stepmod = stepprev;

		hmod = stepmod * h;

		for (k = k; k < (chunk + 1) * sel; k = k+stepmod)
		{	
            max_min_err_exc[0] = max_min_err_exc[1] = false;

			bdwz = bdiffw0 * (k + (butcher_as[0] * stepmod)) * h;// half-step co-ordinate coeff

			NLFN_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_NL,d_A1,d_A1,dt,w0,bdwz,Npoints);
			KN_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A0,d_K2,d_K2,d_NL,d_G,w0,hmod,k,invdt2,Npoints,d_A1,butcher_as[1],0.0);	
			
            bdwz = bdiffw0 * (k + (butcher_as[1] * stepmod)) * h;// three-quarter-step co-ordinate coeff

            NLFN_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_NL,d_A1,d_A1,dt,w0,bdwz,Npoints);
            KN_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A0,d_K3,d_K2,d_NL,d_G,w0,hmod,k,invdt2,Npoints,d_E,butcher_as[2],butcher_as[3]);	
             
			complex_AddVec<<<blocksPerGrid, threadsPerBlock>>>(d_E,d_E,d_K1,butcher_as[4],Npoints);

            bdwz = bdiffw0 * (k + stepmod) * h;// full-step co-ordinate coeff

            NLFN_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_NL,d_E,d_E,dt,w0,bdwz,Npoints);
			KN_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A0,d_K4,d_K3,d_NL,d_G,w0,hmod,k,invdt2,Npoints,d_A1,butcher_bs[0],butcher_bs[1]);
             
			complex_AddVec<<<blocksPerGrid, threadsPerBlock>>>(d_A1,d_A1,d_K2,butcher_bs[2],Npoints);
			complex_AddVec<<<blocksPerGrid, threadsPerBlock>>>(d_A1,d_A1,d_K1,butcher_bs[3],Npoints);

            hipDeviceSynchronize();
			//Error_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_K1, d_K2, d_E, k, stepmod, Npoints, 
			Error_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A1, d_E, d_E, k, stepmod, Npoints, 
																max_pct_err, min_pct_err, max_min_err_exc);															
			//max_min_err_exc[1] = true;															
			hipDeviceSynchronize();													
			
			if (max_min_err_exc[0] && stepmod > 1 && (k - stepmod) > 0)
			{
				k = k - stepmod;
				stepmod = stepmod - 1;
				hmod 	= stepmod * h;
				stepprev = stepmod;
				
				bdwz = bdiffw0 * (k + stepmod) * h;

    			NLFN_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_NL,d_A0,d_A0,dt,w0,bdwz,Npoints);
				KN_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A0,d_K1,d_K1,d_NL,d_G,w0,hmod,k,invdt2,Npoints,d_A1,butcher_as[0],0.0);
				//complex_AddVec<<<blocksPerGrid, threadsPerBlock>>>(d_A1,d_A0,d_K1,butcher_as[0],Npoints);
			}
			else 
			{
				complex_ScaleVec<<<blocksPerGrid, threadsPerBlock>>>(d_E, 1, Npoints, gi1, gi2);
	
				// Transform signal
				hipfftExecC2C(plan, (hipfftComplex *) d_E,
								(hipfftComplex *) d_E,
									HIPFFT_FORWARD);						

				//Dispersion_kernel<<<Npoints/threadsPerBlock, threadsPerBlock>>>(d_Ap+(i * Nchunks), d_E+i, d_beta_op, Npoints, stepmod, chunk, Nchunks);	
				Dispersion_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_Ap, d_E, d_alpha_w, d_beta_op, Npoints, stepmod, chunk, Nchunks, d_stepmods, k);
				//hipDeviceSynchronize();

				// Transform signal back
				hipfftExecC2C(plan, (hipfftComplex *) d_E,
								(hipfftComplex *) d_A0,
									HIPFFT_BACKWARD);

				complex_ScaleVec<<<blocksPerGrid, threadsPerBlock>>>(d_A0, invSize, Npoints, 0, 0); 
/*
                */

                d_temp = d_K1;
                d_K1 = d_K4;
                d_K4 = d_temp;
				hipDeviceSynchronize();
                complex_AddVec<<<blocksPerGrid, threadsPerBlock>>>(d_A1,d_A0,d_K1,butcher_as[0],Npoints);
 				
				
				if (k + stepmod >= (chunk + 1) * sel && k < ((chunk + 1) * sel) - 1)
				{
					
					stepend = (((chunk + 1) * sel) - k) - 1; 
					hmod 	= stepend * h;
					stepprev = stepmod;
					stepmod = stepend;
				}
				else
				{
					if (!max_min_err_exc[1] && !max_min_err_exc[0] && (k + stepmod + stepmod) < ((chunk + 1) * sel) - 1)
					{
						stepmod = stepmod * 2;
						//stepmod = stepmod + 1;
						hmod 	= stepmod * h;
					}
					
					stepprev = stepmod;

				}
			

			} // dispersion
			
		
		} // chunk
		hipDeviceSynchronize();

	} // crystal		
	hipDeviceSynchronize();

  	// Destroy CUFFT context
	hipfftDestroy(plan);

	// Release resources
	hipHostFree(max_min_err_exc);
	//hipFree(max_min_err_exc);

}