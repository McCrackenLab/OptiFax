#include "hip/hip_runtime.h"
/* 
 *OPO_CUDA_DYNAMIC_BATCH_OP.cu
 *
 */

#include <hip/hip_runtime_api.h> 
#include <cuda_device_runtime_api.h> 
#include <hip/hip_math_constants.h>
#include <hipfft/hipfft.h>
#include "OPO_BATCH_HEADER.hpp"
#include <hip/hip_runtime.h>
#include ""

/*Define block size.*/
#define threadsPerBlock	128

/* Define imaginary constant */
#define I make_float2(0.0,1.0)
#define c0 make_float2(0.0,0.0)

__constant__ float w;
__constant__ float d_invdt2;  
__constant__ int Np = 1; 

// complex math functions
__device__ inline
float2 conjugate(float2 arg)
{
    return make_float2(arg.x, -arg.y);
}

__device__ inline
float2 complex_exp(float arg)
{
    return make_float2(cosf(arg), sinf(arg));
}

__device__ inline
float2 complex_pow(float2 arg, int n)
{
	return make_float2(cosf(acosf(arg.x) * n), -sinf(asinf(-arg.y)*n));
}

__device__ inline
float2 complex_add(float2 const a, float2 b)
{
    return make_float2(a.x + b.x, a.y + b.y);
}

__device__ inline
float2 complex_minus(float2 a, float2 b)
{
    return make_float2(a.x - b.x, a.y - b.y);
}

__device__ inline
float2 complex_mult(float2 ab, float2 cd)
{
    return make_float2((ab.x * cd.x) - (ab.y * cd.y), (ab.x * cd.y) + (ab.y * cd.x));
}

__device__ inline
float2 complex_scale(float2 a, float s) 
{
	return make_float2(s * a.x, s * a.y);
}


__host__ __device__ inline
float fabsf2(const float2 &a) 
{ 
	return hypot( a.x , a.y );
}


__device__ inline
float2 f2absf2(const float2 &a) 
{ 
	return make_float2(fabsf2(a),0.0); 
}

__global__
 void complex_AddVec(	float2 * const A,
             			float2 const * const B,
 						int const N)
{
	
    /* Calculate the global linear index, assuming a 1-d grid. */
    /*
    */
    int const gi = blockDim.x * blockIdx.x + threadIdx.x;
    if (gi < N) 
    {
	
        A[gi] = complex_add(A[gi] , B[gi]);
    }
}

 __global__ 
 void complex_ScaleVec(	float2 * const A,
             			float const b,
             			int const N,
             			int const gi1,
             			int const gi2,
						int gy)
{
    /* Calculate the global linear index, assuming a 1-d grid. */
   	// 2D grid?
    int const gx = blockDim.x * blockIdx.x + threadIdx.x;
	int const gi = (blockDim.x * blockIdx.x) + threadIdx.x + (N * gy);

    float pos, g, bc;
	float a = 24.0;

    if (gx < N) 
    {
    	
    	if (gx > gi1 && gx < gi2) // Remove temporal aliasing     		
    	{
    		pos = (a*2.0/N) * (fabs( (float) gx - (N/2.0) ));
        	g = coshf(pos);
    		bc = (1 - (1/(g*g))) * b;
    		A[gi] = complex_scale(A[gi] , bc);
    	}
    	else
    	{
			A[gi] = complex_scale(A[gi] , b);
    	}
    	
    }
}

// Simple all in one NLFN kernel to start with.
/* NLfn = ((A0.^2).*exp(i(wt-(bdiffw0)*z)))
 * +(2*expi(-wt+(bdiffw0)*z).*abs(A0).^2)
 */
__global__
void NLFN_kernel(float2 * const NL, 
				 float2 const * const Ax,
				 float2 const * const Ay,
				 double const dt,
				 float const w0, 
				 double bdwz,
				 int Npoints,
                 int gy)
{

	float arg;
	/* Calculate the global linear index, assuming a 1-d grid. */
	// 2D grid?
    int const gx = blockDim.x * blockIdx.x + threadIdx.x;
	int const gi = (blockDim.x * blockIdx.x) + threadIdx.x + (Npoints * gy);
    if (gx < Npoints)
    {
		/* NL[gi] = ((A0[gi] * A0[gi]) * (expfi(wt[gi] - (bdiffw0 * z)))) +
    				(2 * expfi(-wt[gi] + (bdiffw0 * z)) * (abs(A0[gi]) * abs(A0[gi]) )); */
		if (gx > (Npoints/2))
		{
			arg = ((w0 * (dt*(gx-Npoints/2))) - bdwz);
		}
		else
		{
			arg = ((w0 * (dt*(gx+Npoints/2))) - bdwz);
		}

    	float2 AxAy		= complex_mult(Ax[gi],Ay[gi]);
    	float2 abAxAy 	= complex_mult(f2absf2(Ax[gi]),f2absf2(Ay[gi]));
    	float2 expon 	= complex_exp(arg);
    	float2 dbcnjex	= complex_add((conjugate(expon)),(conjugate(expon))); 
    	
    	NL[gi] 			= complex_add(complex_mult(AxAy, expon), complex_mult(dbcnjex,abAxAy));

    }

}

// Runge-Kutta KN Step Kernel
// K1=-h*1i*G*(NLfn-1/(2*dt)*(1i/w0)*(NLfn(ind1)-NLfn(ind2)))
__global__
void KN_kernel (float2 const * const A0,
				float2 * const KN,
				float2 * const KN2,
				float2 * const NL,
				float const * const G,
				float const w0,
				float const h,
				int const k,
				float invdt2,
				int Npoints,
				float2 * const AN,
				float fracstep,
                int gy)
{
	int in1, in2;

	/* Calculate the global linear index, assuming a 1-d grid. */
 	// 2D grid?
    int const gx = blockDim.x * blockIdx.x + threadIdx.x;
	int const gi = (blockDim.x * blockIdx.x) + threadIdx.x + (Npoints * gy);
    
    int const d = 1;

	float2 iby2dtw0 = complex_mult(make_float2(invdt2/(d * w0), 0.0), I);
    float2 hiG = complex_mult(I, make_float2((-1.0 * h * G[k]),0.0));

    // if cell is not a boundary
    if (gx > d - 1 && gx < Npoints - d)
    {
		in1 = gi+d;
	    in2 = gi-d;
	}   
	else
	{
	    if (gx < d)
	    {
	    in1 = gi+d;
	    in2 = Npoints - d + gi;
	    }
	    if (gx > Npoints - (d + 1))
    	{
    	in1 = gi - (Npoints - d);
    	in2 = gi-d;
	    }
	}

	float2 dNL = complex_minus(NL[in1], NL[in2]);
	KN[gi] = complex_mult(hiG, (complex_minus(NL[gi], complex_mult(iby2dtw0, dNL))));
	 AN[gi] = complex_add(A0[gi],complex_add(complex_scale(KN[gi],fracstep),complex_scale(KN2[gi],fracstep)));
}

__global__ void Dispersion_kernel (	float2 * const ApFT,
									float2 * const EFT,
									float const * const alpha_w,
									float const * const beta_op,
									int Npoints,
									int stepmod,
									int chunk,
									int Nchunks,
									float * const stepmods,
									int const k,
									int gy)									
{
    //int sc;
    float2 bophmod = make_float2(1.0,0.0);

	/* Calculate the global linear index, assuming a 2-d grid. */
    int const gx = blockDim.x * blockIdx.x + threadIdx.x;
	//int const gy = blockIdx.y;
	int const gi = (blockDim.x * blockIdx.x) + threadIdx.x + (Npoints * gy);

	if (gx == k) // Ensure this is performed by a single thread.. maybe gi for batch?
	{
		stepmods[k] = stepmod;
	}

    if (gx < Npoints)
    {
    	bophmod = complex_exp(-beta_op[gx] * stepmod);
		bophmod = complex_scale(bophmod, powf(alpha_w[gx],stepmod));
    	EFT[gi] = complex_mult(EFT[gi],bophmod);
        ApFT[(gy * (Nchunks-1) * Npoints) + (chunk * Npoints) + gi] = EFT[gi];
    }

}
									
__global__ void Error_kernel (	float2 * const K1,
								float2 * const K2,
								float2 * const E,
								int k,
								int stepmod,
								int Npoints,
								float const maxpe,
								float const minpe,
								bool * max_min_flag,
                                int gy)
{
    float errfv, pcterrfv;

  	/* Calculate the global linear index, assuming a 2-d grid. */
    int const gx = blockDim.x * blockIdx.x + threadIdx.x;
	int const gi = (blockDim.x * blockIdx.x) + threadIdx.x + (Npoints * gy);
    if (gx < Npoints)
    {
    	if(fabsf2(E[gi]) > 0)
    	{
			errfv 	 = (fabsf2(complex_minus(K2[gi],K1[gi])));
	    	pcterrfv = 100.0 * (errfv / (fabsf2(E[gi])+1));

	    	if (pcterrfv >= maxpe)
	    	{
	    		max_min_flag[2*gy] = true;
	    	} 
	    	else
	    	{
		    	if (pcterrfv >= minpe)
		    	{
		    		max_min_flag[(2*gy)+1] = true;
		    	} 
    		}
	 	}
    }

}

__global__ void copyKernel (float2 * Eft,
							float2 * ApFT,
							int Npoints,
							int chunk,
							int Nchunks)
{
	/* Calculate the global linear index, assuming a 1-d grid. */
    //int const gi = blockDim.x * blockIdx.x + threadIdx.x;

    //if (gi < Npoints)
	for (int gi =  blockDim.x * blockIdx.x + threadIdx.x;
		gi < Npoints;
		gi += blockDim.x * gridDim.x)	
    {
    	ApFT[chunk + (gi * Nchunks)] = Eft[gi];
    }

} 


__global__ void Heun_dynamic_kernel (float2 * const A0,
                                     float2 * const A1,
                                     float2 * const E,
                                     float2 * const NL, 				                     
                                     float2 * const K1,
                                     float2 * const K2,
                                     float const * const G,                                                                          
				                     double const dt,
				                     float const w0, 
				                     double const bdiffw0,
				                     int const Npoints,
                                     int const k,
                                     float const h,
                                     float stepmod,
                                     float invdt2,
                                     float fracstep,
                                     dim3 const grid,
                                     float const maxpe,
                                     float const minpe,
                                     bool * max_min_flag,
									 bool * max_min_batch_flag,
									 float2 * const Ap,
									 float const * const alpha_w,
									 float const * const beta_op,
									 int chunk,
									 int Nchunks,
									 float * const stepmods)
{

    int gy = blockIdx.y;
    // Launch one child grid per y block
    if ( blockIdx.x == 0 && threadIdx.x == 0) 
    {
	
		float invSize = 1.0 / Npoints;
		int gi1 = 6.0*(Npoints/16);
		int gi2 = 10.0*(Npoints/16);
		int ky;

		hipfftHandle plan;
		hipfftPlan1d(&plan, Npoints, HIPFFT_C2C, 1);

			double bdwz     = bdiffw0 * k * h;
    		double bdwz2    = bdiffw0 * (k + stepmod) * h;
   			float hmod 		= h * stepmod;	
			
		max_min_batch_flag[2*gy] = max_min_batch_flag[(2*gy)+1] = false;

        NLFN_kernel<<<grid.x, threadsPerBlock>>>(NL,A0,A0,dt,w0,bdwz,Npoints,gy);
        KN_kernel<<<grid.x, threadsPerBlock>>>(A0,K1,K1,NL,G,w0,hmod,k,invdt2,Npoints,A1,0.5,gy);

        NLFN_kernel<<<grid.x, threadsPerBlock>>>(NL,A1,A1,dt,w0,bdwz2,Npoints,gy);
        KN_kernel<<<grid.x, threadsPerBlock>>>(A0,K2,K1,NL,G,w0,hmod,k,invdt2,Npoints,E,0.5,gy);

        Error_kernel<<<grid.x, threadsPerBlock>>>(A1,E,E,k,stepmod,Npoints,maxpe,minpe,max_min_batch_flag,gy);
		

		complex_ScaleVec<<<grid.x, threadsPerBlock>>>(E, 1, Npoints, gi1, gi2, gy);
	
				// Transform signal
				hipfftExecC2C(plan, (hipfftComplex *) E,
								   (hipfftComplex *) E,
									HIPFFT_FORWARD);						

		Dispersion_kernel<<<grid.x, threadsPerBlock>>>(Ap,E,alpha_w,beta_op,Npoints,stepmod,chunk,Nchunks,stepmods,k,gy);
				//hipDeviceSynchronize();

				// Transform signal back
				hipfftExecC2C(plan, (hipfftComplex *) E,
								   (hipfftComplex *) A0,
									HIPFFT_BACKWARD);

		complex_ScaleVec<<<grid.x, threadsPerBlock>>>(A0, invSize, Npoints, 0, 0, gy);
				
	}


		/*
		if (max_min_batch_flag[2*gy])
		{
			max_min_flag[0] = true;
		}
		else
		{
			if (max_min_batch_flag[(2*gy)+1])
			{
				max_min_flag[1] = true;
			}
		}
		*/
}




// Host function called by MEX gateway.
 void OPO_TEST_CUDA(float2* const d_E,
					float2 * const d_A0,
					float2 * const d_A1,
					float2 * const d_NL,
					float2 * const d_K1,
					float2 * const d_K2,  
				   	float const * const d_G,
					float const * const d_alpha_w,
				   	float const w0, 
				   	double const bdiffw0,
				   	float const h,
				   	int const Nsteps,
				   	int Npoints,
					int Nbatches,
				   	double const dt,
					float const * const d_beta_op,
					float const max_pct_err,
					float const min_pct_err,
					float const sel,
					float2 * const d_Ap,
					float * const d_stepmods)
{	
	// GPU variables
	float stepmod, stepend, stepprev, k, invdt2, invSize, hmod;
	double bdwz;
	int  chunk, Nchunks, gi1, gi2;
	bool *max_min_err_exc, *max_min_batch_err_exc;

	// Precompute scalars:
	invdt2 	= 1.0 / (2.0 * dt);
	invSize = 1.0 / Npoints;
	k = 0.0;
	stepprev = 4.0;
	Nchunks = Nsteps / sel;
	if (Nchunks*sel < Nsteps)
	{
		Nchunks = Nchunks + 1;
	}
	// Constants defining cut off for temporal aliasing reduction
	gi1 = 6.0*(Npoints/16);
	gi2 = 10.0*(Npoints/16);

	hipHostMalloc(&max_min_err_exc, 2*sizeof(bool));
	hipHostMalloc(&max_min_batch_err_exc, 2*Nbatches*sizeof(bool));


	// CUFFT plan simple API
  	hipfftHandle plan;
  	hipfftPlan1d(&plan, Npoints, HIPFFT_C2C, Nbatches);
	//hipfftPlanMany(&plan, 1, Npoints, HIPFFT_C2C, 1);

	dim3 const blocksPerGrid(((Npoints) + threadsPerBlock - 1) / threadsPerBlock, Nbatches);

	for (chunk = 0; chunk < Nchunks; chunk++)
	{
		stepmod = stepprev;

		hmod = stepmod * h;

		for (k = k; k < (chunk + 1) * sel && k < Nsteps; k = k+stepmod)
		{	
		/*	
		*/
			max_min_err_exc[0] = max_min_err_exc[1] = false;

            Heun_dynamic_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A0,d_A1,d_E,d_NL,d_K1,d_K2,d_G,dt,w0,bdiffw0,Npoints,k,h,stepmod,invdt2,0.5,blocksPerGrid,
                                                                    max_pct_err, min_pct_err, max_min_err_exc, max_min_batch_err_exc,
																	d_Ap,d_alpha_w,d_beta_op,chunk,Nchunks,d_stepmods);
			hipDeviceSynchronize();													
			
			/* 
			
			if (max_min_err_exc[0] && stepmod > 1 && (k - stepmod) > 0)
			{
				k = k - stepmod;
				stepmod = stepmod - 1;
				hmod 	= stepmod * h;
				stepprev = stepmod; 
			}
			else 
			{
				complex_ScaleVec<<<blocksPerGrid, threadsPerBlock>>>(d_E, 1, Npoints, gi1, gi2);
	
				// Transform signal
				hipfftExecC2C(plan, (hipfftComplex *) d_E,
								(hipfftComplex *) d_E,
									HIPFFT_FORWARD);						

				//Dispersion_kernel<<<Npoints/threadsPerBlock, threadsPerBlock>>>(d_Ap+(i * Nchunks), d_E+i, d_beta_op, Npoints, stepmod, chunk, Nchunks);	
				Dispersion_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_Ap, d_E, d_alpha_w, d_beta_op, Npoints, stepmod, chunk, Nchunks, d_stepmods, k);
				//hipDeviceSynchronize();

				// Transform signal back
				hipfftExecC2C(plan, (hipfftComplex *) d_E,
								(hipfftComplex *) d_A0,
									HIPFFT_BACKWARD);

				complex_ScaleVec<<<blocksPerGrid, threadsPerBlock>>>(d_A0, invSize, Npoints, 0, 0);
				
			*/

				if (k + stepmod >= (chunk + 1) * sel && k < ((chunk + 1) * sel) - 1)
				{
					
					stepend = (((chunk + 1) * sel) - k) - 1; 
					hmod 	= stepend * h;
					stepprev = stepmod;
					stepmod = stepend;
				}
				else
				{
					if (!max_min_err_exc[1] && !max_min_err_exc[0] && (k + stepmod + 1) < ((chunk + 1) * sel) - 1)
					{
						//stepmod = stepmod * 2;
						stepmod = stepmod + 1;
						hmod 	= stepmod * h;
					}
					
					stepprev = stepmod;

				}
			
			
			//} // dispersion
			
		
		} // chunk
		hipDeviceSynchronize();

	} // crystal		
	hipDeviceSynchronize();

  	// Destroy CUFFT context
	hipfftDestroy(plan);

	// Release resources
	hipHostFree(max_min_err_exc);
	hipHostFree(max_min_batch_err_exc);


}